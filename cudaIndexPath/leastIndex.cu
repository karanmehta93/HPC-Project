#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include "gputimer.h"

#define MAX_THREADS 1024
using namespace std;
GpuTimer timer;

__global__ void getShortestPath(int *leastEnergySum, int * indexLeastEnergySumInRow, const int height, const int width) {

	/*printf("d1: %d \n", leastEnergySum[1638304]);
	printf("d2: %d \n", leastEnergySum[1638399]);

	if (leastEnergySum[1638304] <= leastEnergySum[1638399]) {
		printf("correct1 \n");
	}

	if (leastEnergySum[1638399] <= leastEnergySum[1638304]) {
		printf("correct2 \n");
	}*/

	int indexLeastEnergySumPath;
	int leastEnergySumPathValue = INT_MAX;
	for (int i = width * (height - 1); i < height * width; i++) {
		if (leastEnergySumPathValue >= leastEnergySum[i]) {
			leastEnergySumPathValue = leastEnergySum[i];
			indexLeastEnergySumPath = i;
		}
	}

	int tempCountHeight = height - 1;
	int tempIndexLeastEnergySumPath = indexLeastEnergySumPath;
	indexLeastEnergySumInRow[tempCountHeight] = indexLeastEnergySumPath;
	while (tempCountHeight > 0) {
		int left = tempIndexLeastEnergySumPath - width - 1;
		int middle = tempIndexLeastEnergySumPath - width;
		int right = tempIndexLeastEnergySumPath - width + 1;
		int leftLimit = (tempCountHeight - 1) * width;
		int rightLimit = tempCountHeight * width;
		if (leftLimit <= left && right < rightLimit) {
			if (leastEnergySum[left] <= leastEnergySum[middle] && leastEnergySum[left] <= leastEnergySum[right])
				tempIndexLeastEnergySumPath = left;
			else if (leastEnergySum[middle] <= leastEnergySum[left] && leastEnergySum[middle] <= leastEnergySum[right])
				tempIndexLeastEnergySumPath = middle;
			else if (leastEnergySum[right] <= leastEnergySum[middle] && leastEnergySum[right] <= leastEnergySum[left])
				tempIndexLeastEnergySumPath = right;
		} else if (leftLimit > left) {
			tempIndexLeastEnergySumPath = leastEnergySum[middle] <= leastEnergySum[right] ? middle : right;
		} else {
			tempIndexLeastEnergySumPath = leastEnergySum[left] <= leastEnergySum[middle] ? left : middle;
		}
		tempCountHeight--;
		indexLeastEnergySumInRow[tempCountHeight] = tempIndexLeastEnergySumPath;
	}
}

int main(int argc, char** argv) {

	int height = 0, width = 0;
	FILE* docRead = fopen("pathIndex", "rb");
	if (!docRead) {
		cout << "pathIndex file not found" << endl;
	}
	fread(&height, sizeof(int), 1, docRead);
	int* testIndexPath = (int *) malloc(height * sizeof(int));
	fread(testIndexPath, sizeof(int), height, docRead);
	fclose(docRead);

	FILE* docRead3 = fopen("minEnergy", "rb");
	if (!docRead3) {
		cout << "minEnergy file not found" << endl;
	}
	fread(&height, sizeof(int), 1, docRead3);
	fread(&width, sizeof(int), 1, docRead3);
	int * h_minEnergy = (int *) malloc(width * height * sizeof(int));
	fread(h_minEnergy, sizeof(int), width * height, docRead3);
	fclose(docRead3);

	hipError_t rc;
	int *d_minEnergy;
	int *h_shortestPath, *d_shortestPath;
	h_shortestPath = (int *) malloc(height * sizeof(int));

	rc = hipMalloc((void**) &d_shortestPath, height * sizeof(int));
	if (rc != hipSuccess) {
		cout << "Malloc Failed for d_shortestPath" << endl;
	}

	rc = hipMalloc((void**) &d_minEnergy, height * width * sizeof(int));
	if (rc != hipSuccess) {
		cout << "Malloc Failed for d_minEnergy" << endl;
	}

	rc = hipMemcpy(d_minEnergy, h_minEnergy, height * width * sizeof(int), hipMemcpyHostToDevice);
	if (rc != hipSuccess) {
		cout << "Memcpy failed from host to device" << endl;
	}

	timer.Start();
	getShortestPath<<<1,1>>>(d_minEnergy, d_shortestPath, height, width);
	timer.Stop();

	rc = hipMemcpy(h_shortestPath, d_shortestPath, height * sizeof(int), hipMemcpyDeviceToHost);
	if (rc != hipSuccess) {
		cout << "Memcpy failed from device to host with rc:" << rc << endl;
	}

	cout << "time taken for the operation:" << timer.Elapsed() << endl;

	for (int i = height-1; i > 0; i--) {
		if (h_shortestPath[i] != testIndexPath[i]) {
			cout << "error at index:" << i << "|Expected: " << testIndexPath[i] << "|actual: " << h_shortestPath[i] << endl;
			break;
		}
	}

	/*int debug = 1;

	if (debug) {
		cout << h_minEnergy[1638304] << endl;
		cout << h_minEnergy[1638399] << endl;
	}*/



	hipFree(d_shortestPath);
	hipFree(d_minEnergy);
	free(h_minEnergy);
	free(testIndexPath);
	return 0;
}