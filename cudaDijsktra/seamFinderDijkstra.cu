#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include "gputimer.h"

#define N 200
#define imgHeight 200
#define MAX_THREADS 1024
#define fMax 999999
using namespace std;
GpuTimer timer;

struct minPixel{
	float energy;
	int y;
};

__global__ void computeMinEnergyMatrix(float *energy, float *min_energy,int height,int width) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int pos = bid * blockDim.x + tid;
    
    extern __shared__ minPixel shared_minSet[];
    __shared__ int shared_minX;
    __shared__ int shared_minY;
    __shared__ int shared_minEnergy;
    __shared__ float shared_energy[N*N];
    
    //float columnEnergy[imgHeight];
    //float minPathEnergy[imgHeight];
    bool activeNodes[imgHeight];
    int minY = 0;
    int minEnergy = fMax;
    
    for(int i = 0; i < height; i++) {
    	shared_energy[width*i + pos] = energy[width*i + pos];
    	min_energy[width*i+pos] = fMax;
    	activeNodes[i] = false;
    }
    
    min_energy[pos] = energy[pos];
    activeNodes[0] = true;
    shared_minSet[pos].energy = energy[pos];
    shared_minSet[pos].y = 0;
    __syncthreads();
    
    for(int i = 0; i < (height*width)-1; i++) {
    	if(tid == 0) {
    		int tMinX = 0;
    		int tMinEnergy = shared_minSet[tMinX].energy;
    		for(int j = 1; j < width; j++) {
    			if(shared_minSet[j].energy < tMinEnergy) {
    				tMinEnergy = shared_minSet[j].energy;
    				tMinX = j;
    			}
    		}
    		shared_minX = tMinX;
    		shared_minY = shared_minSet[tMinX].y; 	
    		shared_minEnergy = tMinEnergy;
    		//printf("minX:%d, minY:%d \n", shared_minX, shared_minY);
    	}
    	 __syncthreads();
    	 
    	 if(shared_minY == height-1) {
    	    break;
    	 }
    	 
    	 if(shared_minX == pos) {
    	    activeNodes[shared_minY] = false;
    	 }
    	 
    	 if(pos == shared_minX-1 || pos == shared_minX || pos == shared_minX+1) {
    	 	if(min_energy[width*(shared_minY+1)+pos] > shared_minEnergy+shared_energy[(shared_minY+1)*width+pos]) {
    	 		min_energy[width*(shared_minY+1)+pos] = shared_minEnergy+shared_energy[(shared_minY+1)*width+pos];
    	 		activeNodes[shared_minY+1] = true;
    	 	}
    	 }
    	 
    	 minEnergy = fMax;
    	 for(int j = 0; j < height; j++) {
    	    if(activeNodes[j] && min_energy[width*j+pos] < minEnergy) {
    	 		minY = j;
    	 		minEnergy = min_energy[width*j+pos];
    	 	}
    	 }
    	 shared_minSet[pos].energy = minEnergy;
    	 shared_minSet[pos].y = minY;
    	 __syncthreads();
    }
    
    /*for(int i = 0; i < height; i++) {
    	min_energy[width*i+pos] = minPathEnergy[i];
    }*/
    __syncthreads();
}

int main(int argc, char** argv)
{
    int height = N;
    int width = N;
    float *h_energy = (float*) malloc(N * N * sizeof(float));
    float *h_min_energy = (float*) malloc(N * N * sizeof(float));

    cout<<"Original Matrix"<<endl;
    for(int i= 0; i<height; i++) {
        for(int j=0; j<width; j++) {
            h_energy[i * N + j] = (i * N + j) > width/2 ? (i * N +j)%23 : (j * N + i)%23;
            //cout << h_energy[i * N + j] << "  ";
        }
        //cout<<endl;
    }   
    
    cout << "-------------------- Minimum Energy Matrix Calculation Starts --------------------" << endl;

    int noOfBlocks = 1; 
    int noOfThreads = min(MAX_THREADS, width);
    int sharedSize = width * sizeof(minPixel);
    
    cout<<"Blocks: "<<noOfBlocks<<"  Threads: "<<noOfThreads<<" SharedSize: "<<sharedSize<<endl;
    
    // Number to blocks will always stay at 1    
    dim3 grid(noOfBlocks), block(noOfThreads);
    float *d_energy, *d_min_energy;   
    hipError_t rc;
    
    rc = hipMalloc((void**) &d_energy, N * N * sizeof(float));
    if(rc != hipSuccess) {
        cout<<"Malloc Failed for d_energy"<<endl;
    }
    rc = hipMalloc((void**) &d_min_energy, N * N * sizeof(float));
    if(rc != hipSuccess) {
        cout<<"Malloc Failed for d_min_energy"<<endl;
    }
    rc = hipMemcpy(d_energy, h_energy, N * N * sizeof(float), hipMemcpyHostToDevice);
    if(rc != hipSuccess) {
        cout<<"Memcpy failed from host to device"<<endl;
    }
    
    timer.Start();
    computeMinEnergyMatrix<<<grid, block, sharedSize>>>(d_energy, d_min_energy, height, width);
    timer.Stop();
    
    rc = hipMemcpy(h_min_energy, d_min_energy, N * N * sizeof(float), hipMemcpyDeviceToHost);
    if(rc != hipSuccess) {
        cout<<"Memcpy failed from device to host"<<endl;
    }
    
    cout<<"Output Matrix"<<endl;
    for(int i=height-1; i< height; i++) {
        for(int j=0; j<width; j++) {
            cout << h_min_energy[i * N + j] << "  ";
        }
        cout<<endl;
    }
    hipFree(d_energy);
    hipFree(d_min_energy);
    cout<<"Time: "<<timer.Elapsed()<<endl;
    fflush(stdout);
    
    cout << "---------------------- Minimum Value Calculation Starts ----------------------" << endl;
    
    timer.Start();
    float mn = h_min_energy[0];
    for(int i=height-1; i<width; i++) {
        if(h_min_energy[i] < mn) {
            mn = h_min_energy[i];
        }
    }
    timer.Stop();
    cout<<"Timer Elapsed: "<<timer.Elapsed()<<endl;
    cout<<"mn: "<<mn<<endl;
    
    
    return 0;
}